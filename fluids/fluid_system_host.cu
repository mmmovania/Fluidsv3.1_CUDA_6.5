#include "hip/hip_runtime.h"
/*
  FLUIDS v.3 - SPH Fluid Simulator for CPU and GPU
  Copyright (C) 2012-2013. Rama Hoetzlein, http://fluids3.com

  Attribute-ZLib license (* See additional part 4)

  This software is provided 'as-is', without any express or implied
  warranty. In no event will the authors be held liable for any damages
  arising from the use of this software.

  Permission is granted to anyone to use this software for any purpose,
  including commercial applications, and to alter it and redistribute it
  freely, subject to the following restrictions:

  1. The origin of this software must not be misrepresented; you must not
     claim that you wrote the original software.
  2. Altered source versions must be plainly marked as such, and must not be
     misrepresented as being the original software.
  3. This notice may not be removed or altered from any source distribution.
  4. Any published work based on this code must include public acknowledgement
     of the origin. This includes following when applicable:
	   - Journal/Paper publications. Credited by reference to work in text & citation.
	   - Public presentations. Credited in at least one slide.
	   - Distributed Games/Apps. Credited as single line in game or app credit page.	 
	 Retaining this additional license term is required in derivative works.
	 Acknowledgement may be provided as:
	   Publication version:  
	      2012-2013, Hoetzlein, Rama C. Fluids v.3 - A Large-Scale, Open Source
	 	  Fluid Simulator. Published online at: http://fluids3.com
	   Single line (slides or app credits):
	      GPU Fluids: Rama C. Hoetzlein (Fluids v3 2013)

 Notes on Clause 4:
  The intent of this clause is public attribution for this contribution, not code use restriction. 
  Both commerical and open source projects may redistribute and reuse without code release.
  However, clause #1 of ZLib indicates that "you must not claim that you wrote the original software". 
  Clause #4 makes this more specific by requiring public acknowledgement to be extended to 
  derivative licenses. 

*/

#include <conio.h>
//#include <cutil.h>				// cutil32.lib
#include <cutil_math.h>				// cutil32.lib
#include <string.h>
#include <assert.h>

#include <windows.h>

//#include <cuda_gl_interop.h>
#include <stdio.h>
#include <math.h>

extern void app_printf ( char* format, ... );
extern void app_printEXIT ( char* format, ... );
extern char app_getch ();

#include "fluid_system_host.cuh"		
#include "fluid_system_kern.cuh"

FluidParams		fcuda;		// CPU Fluid params
FluidParams*	mcuda;		// GPU Fluid params

bufList			fbuf;		// GPU Particle buffers

bool cudaCheck ( hipError_t status, char* msg )
{
	if ( status != hipSuccess ) {
		app_printf ( "CUDA ERROR: %s\n", hipGetErrorString ( status ) );
		app_getch ();

		MessageBox ( NULL, hipGetErrorString ( status), msg, MB_OK );
		return false;
	} else {
		//app_printf ( "%s. OK.\n", msg );
	}
	return true;
}


void cudaExit ()
{
	int argc = 1;	
	char* argv[] = {"fluids"};

	hipDeviceReset();
}

// Initialize CUDA
void cudaInit()
{   
	int argc = 1;
	char* argv[] = {"fluids"};

	int count = 0;
	int i = 0;

	hipError_t err = hipGetDeviceCount(&count);
	if ( err==hipErrorInsufficientDriver) { app_printEXIT( "CUDA driver not installed.\n"); }
	if ( err==hipErrorNoDevice) { app_printEXIT ( "No CUDA device found.\n"); }
	if ( count == 0) { app_printEXIT ( "No CUDA device found.\n"); }

	for(i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		if(hipGetDeviceProperties(&prop, i) == hipSuccess)
			if(prop.major >= 1) break;
	}
	if(i == count) { app_printEXIT ( "No CUDA device found.\n");  }
	hipSetDevice(i);

	app_printf( "CUDA initialized.\n");
 
	hipDeviceProp_t p;
	hipGetDeviceProperties ( &p, 0);
	
	app_printf ( "-- CUDA --\n" );
	app_printf ( "Name:       %s\n", p.name );
	app_printf ( "Revision:   %d.%d\n", p.major, p.minor );
	app_printf ( "Global Mem: %d\n", p.totalGlobalMem );
	app_printf ( "Shared/Blk: %d\n", p.sharedMemPerBlock );
	app_printf ( "Regs/Blk:   %d\n", p.regsPerBlock );
	app_printf ( "Warp Size:  %d\n", p.warpSize );
	app_printf ( "Mem Pitch:  %d\n", p.memPitch );
	app_printf ( "Thrds/Blk:  %d\n", p.maxThreadsPerBlock );
	app_printf ( "Const Mem:  %d\n", p.totalConstMem );
	app_printf ( "Clock Rate: %d\n", p.clockRate );	

	fbuf.mgridactive = 0x0;
	
	// Allocate the sim parameters
	cudaCheck ( hipMalloc ( (void**) &mcuda, sizeof(FluidParams) ),		"Malloc FluidParams mcuda" );

	// Allocate particle buffers
	cudaCheck ( hipMalloc ( (void**) &fbuf.mpos, sizeof(float)*3 ),		"Malloc mpos" );	
	cudaCheck ( hipMalloc ( (void**) &fbuf.mvel, sizeof(float)*3),			"Malloc mvel" );	
	cudaCheck ( hipMalloc ( (void**) &fbuf.mveleval, sizeof(float)*3),		"Malloc mveleval"  );	
	cudaCheck ( hipMalloc ( (void**) &fbuf.mforce, sizeof(float)*3),		"Malloc mforce"  );	
	cudaCheck ( hipMalloc ( (void**) &fbuf.mpress, sizeof(float) ),		"Malloc mpress"  );	
	cudaCheck ( hipMalloc ( (void**) &fbuf.mdensity, sizeof(float) ),		"Malloc mdensity"  );	
	cudaCheck ( hipMalloc ( (void**) &fbuf.mgcell, sizeof(uint)),			"Malloc mgcell"  );	
	cudaCheck ( hipMalloc ( (void**) &fbuf.mgndx, sizeof(uint)),			"Malloc mgndx"  );	
	cudaCheck ( hipMalloc ( (void**) &fbuf.mclr, sizeof(uint)),			"Malloc mclr"  );	

	cudaCheck ( hipMalloc ( (void**) &fbuf.msortbuf, sizeof(uint) ),		"Malloc msortbu" );	

	cudaCheck ( hipMalloc ( (void**) &fbuf.mgrid, 1 ),						"Malloc mgrid"  );
	cudaCheck ( hipMalloc ( (void**) &fbuf.mgridcnt, 1 ),					"Malloc mgridcnt"  );
	cudaCheck ( hipMalloc ( (void**) &fbuf.mgridoff, 1 ),					"Malloc mgridoff" );	
	cudaCheck ( hipMalloc ( (void**) &fbuf.mgridactive, 1 ),				"Malloc mgridactive");

	//cudaCheck ( hipMalloc ( (void**) &fbuf.mcluster, sizeof(uint) ) );	

	preallocBlockSumsInt ( 1 );
};
	
// Compute number of blocks to create
int iDivUp (int a, int b) {
    return (a % b != 0) ? (a / b + 1) : (a / b);
}
void computeNumBlocks (int numPnts, int maxThreads, int &numBlocks, int &numThreads)
{
    numThreads = min( maxThreads, numPnts );
    numBlocks = iDivUp ( numPnts, numThreads );
}

void FluidClearCUDA ()
{
	cudaCheck ( hipFree ( fbuf.mpos ),			"Free mpos" );	
	cudaCheck ( hipFree ( fbuf.mvel ),			"Free mvel" );	
	cudaCheck ( hipFree ( fbuf.mveleval ),		"Free mveleval" );	
	cudaCheck ( hipFree ( fbuf.mforce ),		"Free mforce" );	
	cudaCheck ( hipFree ( fbuf.mpress ),		"Free mpress");	
	cudaCheck ( hipFree ( fbuf.mdensity ),		"Free mdensity" );		
	cudaCheck ( hipFree ( fbuf.mgcell ),		"Free mgcell" );	
	cudaCheck ( hipFree ( fbuf.mgndx ),		"Free mgndx" );	
	cudaCheck ( hipFree ( fbuf.mclr ),			"Free mclr" );	
	//cudaCheck ( hipFree ( fbuf.mcluster ) );	

	cudaCheck ( hipFree ( fbuf.msortbuf ),		"Free msortbuf" );	

	cudaCheck ( hipFree ( fbuf.mgrid ),		"Free mgrid" );
	cudaCheck ( hipFree ( fbuf.mgridcnt ),		"Free mgridcnt" );
	cudaCheck ( hipFree ( fbuf.mgridoff ),		"Free mgridoff" );
	cudaCheck ( hipFree ( fbuf.mgridactive ),	"Free mgridactive" );
}


void FluidSetupCUDA ( int num, int gsrch, int3 res, float3 size, float3 delta, float3 gmin, float3 gmax, int total, int chk )
{	
	fcuda.pnum = num;	
	fcuda.gridRes = res;
	fcuda.gridSize = size;
	fcuda.gridDelta = delta;
	fcuda.gridMin = gmin;
	fcuda.gridMax = gmax;
	fcuda.gridTotal = total;
	fcuda.gridSrch = gsrch;
	fcuda.gridAdjCnt = gsrch*gsrch*gsrch;
	fcuda.gridScanMax = res;
	fcuda.gridScanMax -= make_int3( fcuda.gridSrch, fcuda.gridSrch, fcuda.gridSrch );
	fcuda.chk = chk;

	// Build Adjacency Lookup
	int cell = 0;
	for (int y=0; y < gsrch; y++ ) 
		for (int z=0; z < gsrch; z++ ) 
			for (int x=0; x < gsrch; x++ ) 
				fcuda.gridAdj [ cell++]  = ( y * fcuda.gridRes.z+ z )*fcuda.gridRes.x +  x ;			
	
	app_printf ( "CUDA Adjacency Table\n");
	for (int n=0; n < fcuda.gridAdjCnt; n++ ) {
		app_printf ( "  ADJ: %d, %d\n", n, fcuda.gridAdj[n] );
	}	

	// Compute number of blocks and threads
	
	int threadsPerBlock = 192;

    computeNumBlocks ( fcuda.pnum, threadsPerBlock, fcuda.numBlocks, fcuda.numThreads);				// particles
    computeNumBlocks ( fcuda.gridTotal, threadsPerBlock, fcuda.gridBlocks, fcuda.gridThreads);		// grid cell
    
	// Allocate particle buffers
    fcuda.szPnts = (fcuda.numBlocks  * fcuda.numThreads);     
    app_printf ( "CUDA Allocate: \n" );
	app_printf ( "  Pnts: %d, t:%dx%d=%d, Size:%d\n", fcuda.pnum, fcuda.numBlocks, fcuda.numThreads, fcuda.numBlocks*fcuda.numThreads, fcuda.szPnts);
    app_printf ( "  Grid: %d, t:%dx%d=%d, bufGrid:%d, Res: %dx%dx%d\n", fcuda.gridTotal, fcuda.gridBlocks, fcuda.gridThreads, fcuda.gridBlocks*fcuda.gridThreads, fcuda.szGrid, (int) fcuda.gridRes.x, (int) fcuda.gridRes.y, (int) fcuda.gridRes.z );		
	
	cudaCheck ( hipMalloc ( (void**) &fbuf.mpos,		fcuda.szPnts*sizeof(float)*3 ),	"Malloc mpos" );	
	cudaCheck ( hipMalloc ( (void**) &fbuf.mvel,		fcuda.szPnts*sizeof(float)*3 ),	"Malloc mvel" );	
	cudaCheck ( hipMalloc ( (void**) &fbuf.mveleval,	fcuda.szPnts*sizeof(float)*3 ),	"Malloc mveleval" );	
	cudaCheck ( hipMalloc ( (void**) &fbuf.mforce,	fcuda.szPnts*sizeof(float)*3 ),		"Malloc mforce" );	
	cudaCheck ( hipMalloc ( (void**) &fbuf.mpress,	fcuda.szPnts*sizeof(float) ),		"Malloc mpress" );	
	cudaCheck ( hipMalloc ( (void**) &fbuf.mdensity,	fcuda.szPnts*sizeof(float) ),	"Malloc mdensity" );	
	cudaCheck ( hipMalloc ( (void**) &fbuf.mgcell,	fcuda.szPnts*sizeof(uint) ),		"Malloc mgcell" );	
	cudaCheck ( hipMalloc ( (void**) &fbuf.mgndx,		fcuda.szPnts*sizeof(uint)),		"Malloc mgndx" );	
	cudaCheck ( hipMalloc ( (void**) &fbuf.mclr,		fcuda.szPnts*sizeof(uint) ),	"Malloc mclr" );	
	//cudaCheck ( hipMalloc ( (void**) &fbuf.mcluster,	fcuda.szPnts*sizeof(uint) ) );	

	int temp_size = 4*(sizeof(float)*3) + 2*sizeof(float) + 2*sizeof(int) + sizeof(uint);
	cudaCheck ( hipMalloc ( (void**) &fbuf.msortbuf,	fcuda.szPnts*temp_size ),		"Malloc msortbuf" );

	// Allocate grid
	fcuda.szGrid = (fcuda.gridBlocks * fcuda.gridThreads);  
	cudaCheck ( hipMalloc ( (void**) &fbuf.mgrid,		fcuda.szPnts*sizeof(int) ),		"Malloc mgrid" );
	cudaCheck ( hipMalloc ( (void**) &fbuf.mgridcnt,	fcuda.szGrid*sizeof(int) ),		"Malloc mgridcnt" );
	cudaCheck ( hipMalloc ( (void**) &fbuf.mgridoff,	fcuda.szGrid*sizeof(int) ),		"Malloc mgridoff" );
	cudaCheck ( hipMalloc ( (void**) &fbuf.mgridactive, fcuda.szGrid*sizeof(int) ),	"Malloc mgridactive" );
		
	// Transfer sim params to device
	updateSimParams ( &fcuda );
	
	hipDeviceSynchronize ();

	// Prefix Sum - Preallocate Block sums for Sorting
	deallocBlockSumsInt ();
	preallocBlockSumsInt ( fcuda.gridTotal );
}

void FluidParamCUDA ( float ss, float sr, float pr, float mass, float rest, float3 bmin, float3 bmax, float estiff, float istiff, float visc, float damp, float fmin, float fmax, float ffreq, float gslope, float gx, float gy, float gz, float al, float vl )
{
	fcuda.psimscale = ss;
	fcuda.psmoothradius = sr;
	fcuda.pradius = pr;
	fcuda.r2 = sr * sr;
	fcuda.pmass = mass;
	fcuda.prest_dens = rest;	
	fcuda.pboundmin = bmin;
	fcuda.pboundmax = bmax;
	fcuda.pextstiff = estiff;
	fcuda.pintstiff = istiff;
	fcuda.pvisc = visc;
	fcuda.pdamp = damp;
	fcuda.pforce_min = fmin;
	fcuda.pforce_max = fmax;
	fcuda.pforce_freq = ffreq;
	fcuda.pground_slope = gslope;
	fcuda.pgravity = make_float3( gx, gy, gz );
	fcuda.AL = al;
	fcuda.AL2 = al * al;
	fcuda.VL = vl;
	fcuda.VL2 = vl * vl;

	//app_printf ( "Bound Min: %f %f %f\n", bmin.x, bmin.y, bmin.z );
	//app_printf ( "Bound Max: %f %f %f\n", bmax.x, bmax.y, bmax.z );

	fcuda.pdist = pow ( fcuda.pmass / fcuda.prest_dens, 1/3.0f );
	fcuda.poly6kern = 315.0f / (64.0f * 3.141592 * pow( sr, 9.0f) );
	fcuda.spikykern = -45.0f / (3.141592 * pow( sr, 6.0f) );
	fcuda.lapkern = 45.0f / (3.141592 * pow( sr, 6.0f) );	

	fcuda.d2 = fcuda.psimscale * fcuda.psimscale;
	fcuda.rd2 = fcuda.r2 / fcuda.d2;
	fcuda.vterm = fcuda.lapkern * fcuda.pvisc;

	// Transfer sim params to device
	updateSimParams ( &fcuda );

	hipDeviceSynchronize ();
}

void CopyToCUDA ( float* pos, float* vel, float* veleval, float* force, float* pressure, float* density, uint* cluster, uint* gnext, char* clr )
{
	// Send particle buffers
	int numPoints = fcuda.pnum;
	cudaCheck( hipMemcpy ( fbuf.mpos,		pos,			numPoints*sizeof(float)*3, hipMemcpyHostToDevice ), 	"Memcpy mpos ToDev" );	
	cudaCheck( hipMemcpy ( fbuf.mvel,		vel,			numPoints*sizeof(float)*3, hipMemcpyHostToDevice ), 	"Memcpy mvel ToDev" );
	cudaCheck( hipMemcpy ( fbuf.mveleval, veleval,		numPoints*sizeof(float)*3, hipMemcpyHostToDevice ), 		"Memcpy mveleval ToDev"  );
	cudaCheck( hipMemcpy ( fbuf.mforce,	force,			numPoints*sizeof(float)*3, hipMemcpyHostToDevice ), 	"Memcpy mforce ToDev"  );
	cudaCheck( hipMemcpy ( fbuf.mpress,	pressure,		numPoints*sizeof(float),  hipMemcpyHostToDevice ), 	"Memcpy mpress ToDev"  );
	cudaCheck( hipMemcpy ( fbuf.mdensity, density,		numPoints*sizeof(float),  hipMemcpyHostToDevice ), 		"Memcpy mdensity ToDev"  );
	cudaCheck( hipMemcpy ( fbuf.mclr,		clr,			numPoints*sizeof(uint), hipMemcpyHostToDevice ), 		"Memcpy mclr ToDev"  );

	hipDeviceSynchronize ();	
}

void CopyFromCUDA ( float* pos, float* vel, float* veleval, float* force, float* pressure, float* density, uint* cluster, uint* gnext, char* clr )
{
	// Return particle buffers
	int numPoints = fcuda.pnum;
	if ( pos != 0x0 ) cudaCheck( hipMemcpy ( pos,		fbuf.mpos,			numPoints*sizeof(float)*3, hipMemcpyDeviceToHost ),	"Memcpy mpos FromDev"  );
	if ( clr != 0x0 ) cudaCheck( hipMemcpy ( clr,		fbuf.mclr,			numPoints*sizeof(uint),  hipMemcpyDeviceToHost ), 		"Memcpy mclr FromDev"  );
	/*cudaCheck( hipMemcpy ( vel,		fbuf.mvel,			numPoints*sizeof(float)*3, hipMemcpyDeviceToHost ) );
	cudaCheck( hipMemcpy ( veleval,	fbuf.mveleval,		numPoints*sizeof(float)*3, hipMemcpyDeviceToHost ) );
	cudaCheck( hipMemcpy ( force,		fbuf.mforce,		numPoints*sizeof(float)*3, hipMemcpyDeviceToHost ) );
	cudaCheck( hipMemcpy ( pressure,	fbuf.mpress,		numPoints*sizeof(float),  hipMemcpyDeviceToHost ) );
	cudaCheck( hipMemcpy ( density,	fbuf.mdensity,		numPoints*sizeof(float),  hipMemcpyDeviceToHost ) );*/
	
	hipDeviceSynchronize ();	
}


void InsertParticlesCUDA ( uint* gcell, uint* ccell, int* gcnt )
{
	hipMemset ( fbuf.mgridcnt, 0,			fcuda.gridTotal * sizeof(int));

	insertParticles<<< fcuda.numBlocks, fcuda.numThreads>>> ( fbuf, fcuda.pnum );
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		fprintf ( stderr,  "CUDA ERROR: InsertParticlesCUDA: %s\n", hipGetErrorString(error) );
	}  
	hipDeviceSynchronize ();
	// Transfer data back if requested (for validation)
	if (gcell != 0x0) {
		cudaCheck( hipMemcpy ( gcell,	fbuf.mgcell,	fcuda.pnum*sizeof(uint),		hipMemcpyDeviceToHost ),  "Memcpy mgcell FromDev");		
		cudaCheck( hipMemcpy ( gcnt,	fbuf.mgridcnt,	fcuda.gridTotal*sizeof(int),	hipMemcpyDeviceToHost ),  "Memcpy mgridcnt FromDev" );
		//cudaCheck( hipMemcpy ( ccell,	fbuf.mcluster,	fcuda.pnum*sizeof(uint),		hipMemcpyDeviceToHost ) );
	}
	
}

void PrefixSumCellsCUDA ( int* goff )
{
	// Prefix Sum - determine grid offsets
    prescanArrayRecursiveInt ( fbuf.mgridoff, fbuf.mgridcnt, fcuda.gridTotal, 0);
	hipDeviceSynchronize ();

	// Transfer data back if requested
	if ( goff != 0x0 ) {
		cudaCheck( hipMemcpy ( goff,	fbuf.mgridoff, fcuda.gridTotal * sizeof(int),  hipMemcpyDeviceToHost ),  "Memcpy mgoff FromDev" );
	}
}

void CountingSortIndexCUDA ( uint* ggrid )
{	
	// Counting Sort - pass one, determine grid counts
	hipMemset ( fbuf.mgrid,	GRID_UCHAR,	fcuda.pnum * sizeof(int) );

	countingSortIndex <<< fcuda.numBlocks, fcuda.numThreads>>> ( fbuf, fcuda.pnum );		
	hipDeviceSynchronize ();

	// Transfer data back if requested
	if ( ggrid != 0x0 ) {
		cudaCheck( hipMemcpy ( ggrid,	fbuf.mgrid, fcuda.pnum * sizeof(uint), hipMemcpyDeviceToHost ), "Memcpy mgrid FromDev" );
	}
}

void CountingSortFullCUDA ( uint* ggrid )
{
	// Transfer particle data to temp buffers
	int n = fcuda.pnum;
	cudaCheck ( hipMemcpy ( fbuf.msortbuf + n*BUF_POS,		fbuf.mpos,		n*sizeof(float)*3,	hipMemcpyDeviceToDevice ),		"Memcpy msortbuf->mpos DevToDev" );
	cudaCheck ( hipMemcpy ( fbuf.msortbuf + n*BUF_VEL,		fbuf.mvel,		n*sizeof(float)*3,	hipMemcpyDeviceToDevice ),		"Memcpy msortbuf->mvel DevToDev" );
	cudaCheck ( hipMemcpy ( fbuf.msortbuf + n*BUF_VELEVAL,	fbuf.mveleval,	n*sizeof(float)*3,	hipMemcpyDeviceToDevice ),		"Memcpy msortbuf->mveleval DevToDev" );
	cudaCheck ( hipMemcpy ( fbuf.msortbuf + n*BUF_FORCE,	fbuf.mforce,	n*sizeof(float)*3,	hipMemcpyDeviceToDevice ),		"Memcpy msortbuf->mforce DevToDev" );
	cudaCheck ( hipMemcpy ( fbuf.msortbuf + n*BUF_PRESS,	fbuf.mpress,	n*sizeof(float),	hipMemcpyDeviceToDevice ),		"Memcpy msortbuf->mpress DevToDev" );
	cudaCheck ( hipMemcpy ( fbuf.msortbuf + n*BUF_DENS,	fbuf.mdensity,	n*sizeof(float),	hipMemcpyDeviceToDevice ),		"Memcpy msortbuf->mdens DevToDev" );
	cudaCheck ( hipMemcpy ( fbuf.msortbuf + n*BUF_GCELL,	fbuf.mgcell,	n*sizeof(uint),		hipMemcpyDeviceToDevice ),		"Memcpy msortbuf->mgcell DevToDev" );
	cudaCheck ( hipMemcpy ( fbuf.msortbuf + n*BUF_GNDX,	fbuf.mgndx,		n*sizeof(uint),		hipMemcpyDeviceToDevice ),		"Memcpy msortbuf->mgndx DevToDev" );
	cudaCheck ( hipMemcpy ( fbuf.msortbuf + n*BUF_CLR,		fbuf.mclr,		n*sizeof(uint),		hipMemcpyDeviceToDevice ),		"Memcpy msortbuf->mclr DevToDev" );

	// Counting Sort - pass one, determine grid counts
	hipMemset ( fbuf.mgrid,	GRID_UCHAR,	fcuda.pnum * sizeof(int) );

	countingSortFull <<< fcuda.numBlocks, fcuda.numThreads>>> ( fbuf, fcuda.pnum );		
	hipDeviceSynchronize ();
}

void ComputePressureCUDA ()
{
	computePressure<<< fcuda.numBlocks, fcuda.numThreads>>> ( fbuf, fcuda.pnum );	
    hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		fprintf ( stderr, "CUDA ERROR: ComputePressureCUDA: %s\n", hipGetErrorString(error) );
	}    
	hipDeviceSynchronize ();
}
void ComputeQueryCUDA ()
{
	computeQuery<<< fcuda.numBlocks, fcuda.numThreads>>> ( fbuf, fcuda.pnum );	
    hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		fprintf ( stderr, "CUDA ERROR: ComputePressureCUDA: %s\n", hipGetErrorString(error) );
	}    
	hipDeviceSynchronize ();
}

void CountActiveCUDA ()
{
	int threads = 1;
	int blocks = 1;
	
	assert ( fbuf.mgridactive != 0x0 );
	
	/*#ifdef CUDA_42
		hipMemcpyToSymbol(HIP_SYMBOL( "gridActive"), &fcuda.gridActive, sizeof(int) );
	#else
		hipMemcpyToSymbol(HIP_SYMBOL( gridActive), &fcuda.gridActive, sizeof(int) );
	#endif */
	
	countActiveCells<<< blocks, threads >>> ( fbuf, fcuda.gridTotal );
	hipDeviceSynchronize ();

	hipMemcpyFromSymbol( &fcuda.gridActive, HIP_SYMBOL("gridActive"), sizeof(int) );
	
	app_printf ( "Active cells: %d\n", fcuda.gridActive );
}

void ComputePressureGroupCUDA ()
{
	if ( fcuda.gridActive > 0 ) {

		int threads = 128;		// should be based on maximum occupancy
		uint3 blocks;
		blocks.x = 4096;
		blocks.y = (fcuda.gridActive / 4096 )+1;
		blocks.z = 1;

		computePressureGroup<<< blocks, threads >>> ( fbuf, fcuda.pnum );	
		hipError_t error = hipGetLastError();
		if (error != hipSuccess) {
			fprintf ( stderr, "CUDA ERROR: ComputePressureGroupCUDA: %s\n", hipGetErrorString(error) );
		}   
		hipDeviceSynchronize ();
	}
}

void ComputeForceCUDA ()
{
	computeForce<<< fcuda.numBlocks, fcuda.numThreads>>> ( fbuf, fcuda.pnum );
    hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		fprintf ( stderr,  "CUDA ERROR: ComputeForceCUDA: %s\n", hipGetErrorString(error) );
	}    
	hipDeviceSynchronize ();
}

void AdvanceCUDA ( float tm, float dt, float ss )
{
	advanceParticles<<< fcuda.numBlocks, fcuda.numThreads>>> ( tm, dt, ss, fbuf, fcuda.pnum );
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		fprintf ( stderr,  "CUDA ERROR: AdvanceCUDA: %s\n", hipGetErrorString(error) );
	}    
    hipDeviceSynchronize ();
}



/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation and 
 * any modifications thereto.  Any use, reproduction, disclosure, or distribution 
 * of this software and related documentation without an express license 
 * agreement from NVIDIA Corporation is strictly prohibited.
 * 
 */

// includes, kernels
#include <assert.h>

inline bool isPowerOfTwo(int n) { return ((n&(n-1))==0) ; }

inline int floorPow2(int n) {
	#ifdef WIN32
		return 1 << (int)logb((float)n);
	#else
		int exp;
		frexp((float)n, &exp);
		return 1 << (exp - 1);
	#endif
}

#define BLOCK_SIZE 256

float**			g_scanBlockSums = 0;
int**			g_scanBlockSumsInt = 0;
unsigned int	g_numEltsAllocated = 0;
unsigned int	g_numLevelsAllocated = 0;

void preallocBlockSums(unsigned int maxNumElements)
{
    assert(g_numEltsAllocated == 0); // shouldn't be called 

    g_numEltsAllocated = maxNumElements;
    unsigned int blockSize = BLOCK_SIZE; // max size of the thread blocks
    unsigned int numElts = maxNumElements;
    int level = 0;

    do {       
        unsigned int numBlocks =   max(1, (int)ceil((float)numElts / (2.f * blockSize)));
        if (numBlocks > 1) level++;
        numElts = numBlocks;
    } while (numElts > 1);

    g_scanBlockSums = (float**) malloc(level * sizeof(float*));
    g_numLevelsAllocated = level;
    
    numElts = maxNumElements;
    level = 0;
    
    do {       
        unsigned int numBlocks = max(1, (int)ceil((float)numElts / (2.f * blockSize)));
        if (numBlocks > 1) 
			cudaCheck ( hipMalloc((void**) &g_scanBlockSums[level++], numBlocks * sizeof(float)), "Malloc prescanBlockSums g_scanBlockSums");
        numElts = numBlocks;
    } while (numElts > 1);

}
void preallocBlockSumsInt (unsigned int maxNumElements)
{
    assert(g_numEltsAllocated == 0); // shouldn't be called 

    g_numEltsAllocated = maxNumElements;
    unsigned int blockSize = BLOCK_SIZE; // max size of the thread blocks
    unsigned int numElts = maxNumElements;
    int level = 0;

    do {       
        unsigned int numBlocks =   max(1, (int)ceil((float)numElts / (2.f * blockSize)));
        if (numBlocks > 1) level++;
        numElts = numBlocks;
    } while (numElts > 1);

    g_scanBlockSumsInt = (int**) malloc(level * sizeof(int*));
    g_numLevelsAllocated = level;
    
    numElts = maxNumElements;
    level = 0;
    
    do {       
        unsigned int numBlocks = max(1, (int)ceil((float)numElts / (2.f * blockSize)));
        if (numBlocks > 1) cudaCheck ( hipMalloc((void**) &g_scanBlockSumsInt[level++], numBlocks * sizeof(int)), "Malloc prescanBlockSumsInt g_scanBlockSumsInt");
        numElts = numBlocks;
    } while (numElts > 1);
}

void deallocBlockSums()
{
	if ( g_scanBlockSums != 0x0 ) {
		for (unsigned int i = 0; i < g_numLevelsAllocated; i++) 
			cudaCheck ( hipFree(g_scanBlockSums[i]), "Malloc deallocBlockSums g_scanBlockSums");
    
		free( (void**)g_scanBlockSums );
	}

    g_scanBlockSums = 0;
    g_numEltsAllocated = 0;
    g_numLevelsAllocated = 0;
}
void deallocBlockSumsInt()
{
	if ( g_scanBlockSums != 0x0 ) {
		for (unsigned int i = 0; i < g_numLevelsAllocated; i++) 
			cudaCheck ( hipFree(g_scanBlockSumsInt[i]), "Malloc deallocBlockSumsInt g_scanBlockSumsInt");
		free( (void**)g_scanBlockSumsInt );
	}

    g_scanBlockSumsInt = 0;
    g_numEltsAllocated = 0;
    g_numLevelsAllocated = 0;
}



void prescanArrayRecursive (float *outArray, const float *inArray, int numElements, int level)
{
    unsigned int blockSize = BLOCK_SIZE; // max size of the thread blocks
    unsigned int numBlocks = max(1, (int)ceil((float)numElements / (2.f * blockSize)));
    unsigned int numThreads;

    if (numBlocks > 1)
        numThreads = blockSize;
    else if (isPowerOfTwo(numElements))
        numThreads = numElements / 2;
    else
        numThreads = floorPow2(numElements);

    unsigned int numEltsPerBlock = numThreads * 2;

    // if this is a non-power-of-2 array, the last block will be non-full
    // compute the smallest power of 2 able to compute its scan.
    unsigned int numEltsLastBlock = numElements - (numBlocks-1) * numEltsPerBlock;
    unsigned int numThreadsLastBlock = max(1, numEltsLastBlock / 2);
    unsigned int np2LastBlock = 0;
    unsigned int sharedMemLastBlock = 0;
    
    if (numEltsLastBlock != numEltsPerBlock) {
        np2LastBlock = 1;
        if(!isPowerOfTwo(numEltsLastBlock)) numThreadsLastBlock = floorPow2(numEltsLastBlock);            
        unsigned int extraSpace = (2 * numThreadsLastBlock) / NUM_BANKS;
        sharedMemLastBlock = sizeof(float) * (2 * numThreadsLastBlock + extraSpace);
    }

    // padding space is used to avoid shared memory bank conflicts
    unsigned int extraSpace = numEltsPerBlock / NUM_BANKS;
    unsigned int sharedMemSize = sizeof(float) * (numEltsPerBlock + extraSpace);

	#ifdef DEBUG
		if (numBlocks > 1) assert(g_numEltsAllocated >= numElements);
	#endif

    // setup execution parameters
    // if NP2, we process the last block separately
    dim3  grid(max(1, numBlocks - np2LastBlock), 1, 1); 
    dim3  threads(numThreads, 1, 1);

    // execute the scan
    if (numBlocks > 1) {
        prescan<true, false><<< grid, threads, sharedMemSize >>> (outArray, inArray,  g_scanBlockSums[level], numThreads * 2, 0, 0);
        if (np2LastBlock) {
            prescan<true, true><<< 1, numThreadsLastBlock, sharedMemLastBlock >>> (outArray, inArray, g_scanBlockSums[level], numEltsLastBlock, numBlocks - 1, numElements - numEltsLastBlock);
        }

        // After scanning all the sub-blocks, we are mostly done.  But now we 
        // need to take all of the last values of the sub-blocks and scan those.  
        // This will give us a new value that must be added to each block to 
        // get the final results.
        // recursive (CPU) call
        prescanArrayRecursive (g_scanBlockSums[level], g_scanBlockSums[level], numBlocks, level+1);

        uniformAdd<<< grid, threads >>> (outArray, g_scanBlockSums[level], numElements - numEltsLastBlock, 0, 0);
        if (np2LastBlock) {
            uniformAdd<<< 1, numThreadsLastBlock >>>(outArray, g_scanBlockSums[level], numEltsLastBlock, numBlocks - 1, numElements - numEltsLastBlock);
        }
    } else if (isPowerOfTwo(numElements)) {
        prescan<false, false><<< grid, threads, sharedMemSize >>> (outArray, inArray, 0, numThreads * 2, 0, 0);
    } else {
        prescan<false, true><<< grid, threads, sharedMemSize >>> (outArray, inArray, 0, numElements, 0, 0);
    }
}

void prescanArrayRecursiveInt (int *outArray, const int *inArray, int numElements, int level)
{
    unsigned int blockSize = BLOCK_SIZE; // max size of the thread blocks
    unsigned int numBlocks = max(1, (int)ceil((float)numElements / (2.f * blockSize)));
    unsigned int numThreads;

    if (numBlocks > 1)
        numThreads = blockSize;
    else if (isPowerOfTwo(numElements))
        numThreads = numElements / 2;
    else
        numThreads = floorPow2(numElements);

    unsigned int numEltsPerBlock = numThreads * 2;

    // if this is a non-power-of-2 array, the last block will be non-full
    // compute the smallest power of 2 able to compute its scan.
    unsigned int numEltsLastBlock = numElements - (numBlocks-1) * numEltsPerBlock;
    unsigned int numThreadsLastBlock = max(1, numEltsLastBlock / 2);
    unsigned int np2LastBlock = 0;
    unsigned int sharedMemLastBlock = 0;
    
    if (numEltsLastBlock != numEltsPerBlock) {
        np2LastBlock = 1;
        if(!isPowerOfTwo(numEltsLastBlock)) numThreadsLastBlock = floorPow2(numEltsLastBlock);            
        unsigned int extraSpace = (2 * numThreadsLastBlock) / NUM_BANKS;
        sharedMemLastBlock = sizeof(float) * (2 * numThreadsLastBlock + extraSpace);
    }

    // padding space is used to avoid shared memory bank conflicts
    unsigned int extraSpace = numEltsPerBlock / NUM_BANKS;
    unsigned int sharedMemSize = sizeof(float) * (numEltsPerBlock + extraSpace);

	#ifdef DEBUG
		if (numBlocks > 1) assert(g_numEltsAllocated >= numElements);
	#endif

    // setup execution parameters
    // if NP2, we process the last block separately
    dim3  grid(max(1, numBlocks - np2LastBlock), 1, 1); 
    dim3  threads(numThreads, 1, 1);

    // execute the scan
    if (numBlocks > 1) {
        prescanInt <true, false><<< grid, threads, sharedMemSize >>> (outArray, inArray,  g_scanBlockSumsInt[level], numThreads * 2, 0, 0);
        if (np2LastBlock) {
            prescanInt <true, true><<< 1, numThreadsLastBlock, sharedMemLastBlock >>> (outArray, inArray, g_scanBlockSumsInt[level], numEltsLastBlock, numBlocks - 1, numElements - numEltsLastBlock);
        }

        // After scanning all the sub-blocks, we are mostly done.  But now we 
        // need to take all of the last values of the sub-blocks and scan those.  
        // This will give us a new value that must be added to each block to 
        // get the final results.
        // recursive (CPU) call
        prescanArrayRecursiveInt (g_scanBlockSumsInt[level], g_scanBlockSumsInt[level], numBlocks, level+1);

        uniformAddInt <<< grid, threads >>> (outArray, g_scanBlockSumsInt[level], numElements - numEltsLastBlock, 0, 0);
        if (np2LastBlock) {
            uniformAddInt <<< 1, numThreadsLastBlock >>>(outArray, g_scanBlockSumsInt[level], numEltsLastBlock, numBlocks - 1, numElements - numEltsLastBlock);
        }
    } else if (isPowerOfTwo(numElements)) {
        prescanInt <false, false><<< grid, threads, sharedMemSize >>> (outArray, inArray, 0, numThreads * 2, 0, 0);
    } else {
        prescanInt <false, true><<< grid, threads, sharedMemSize >>> (outArray, inArray, 0, numElements, 0, 0);
    }
}


void prescanArray ( float *d_odata, float *d_idata, int num )
{	
	// preform prefix sum
	preallocBlockSums( num );
    prescanArrayRecursive ( d_odata, d_idata, num, 0);
	deallocBlockSums();
}
void prescanArrayInt ( int *d_odata, int *d_idata, int num )
{	
	// preform prefix sum
	preallocBlockSumsInt ( num );
    prescanArrayRecursiveInt ( d_odata, d_idata, num, 0);
	deallocBlockSumsInt ();
}

char* d_idata = NULL;
char* d_odata = NULL;

void prefixSum ( int num )
{
	prescanArray ( (float*) d_odata, (float*) d_idata, num );
}

void prefixSumInt ( int num )
{	
	prescanArrayInt ( (int*) d_odata, (int*) d_idata, num );
}

void prefixSumToGPU ( char* inArray, int num, int siz )
{
    cudaCheck ( hipMalloc( (void**) &d_idata, num*siz ),	"Malloc prefixumSimToGPU idata");
    cudaCheck ( hipMalloc( (void**) &d_odata, num*siz ),	"Malloc prefixumSimToGPU odata" );
    cudaCheck ( hipMemcpy( d_idata, inArray, num*siz, hipMemcpyHostToDevice),	"Memcpy inArray->idata" );
}
void prefixSumFromGPU ( char* outArray, int num, int siz )
{		
	cudaCheck ( hipMemcpy( outArray, d_odata, num*siz, hipMemcpyDeviceToHost), "Memcpy odata->outArray" );
	cudaCheck ( hipFree( d_idata ), "Free idata" );
    cudaCheck ( hipFree( d_odata ), "Free odata" );
	d_idata = NULL;
	d_odata = NULL;
}
